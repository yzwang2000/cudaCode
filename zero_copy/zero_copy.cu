#include <cassert>
#include <chrono>
#include <functional>
#include <iomanip>
#include <iostream>
#include <stdexcept>
#include <thread>
#include <tuple>
#include <utility>
#include <vector>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, int num_repeats = 100,
                          int num_warmups = 100)
{
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (int i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (int i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

__global__ void float_addition(float* output, float const* input_1,
                               float const* input_2, uint32_t n)
{
    const uint32_t idx{blockDim.x * blockIdx.x + threadIdx.x};
    const uint32_t stride{blockDim.x * gridDim.x};
    for (uint32_t i{idx}; i < n; i += stride)
    {
        output[i] = input_1[i] + input_2[i];
    }
}

void launch_float_addition_non_mapped_pinned_memory(
    float* h_output, float const* h_input_1, float const* h_input_2,
    float* d_output, float* d_input_1, float* d_input_2, uint32_t n,
    hipStream_t stream)
{
    // 主机内存拷贝到全局内存中
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_input_1, h_input_1, n * sizeof(float),
                                     hipMemcpyHostToDevice, stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_input_2, h_input_2, n * sizeof(float),
                                     hipMemcpyHostToDevice, stream));

    dim3 const threads_per_block{1024};
    dim3 const blocks_per_grid{32};
    float_addition<<<blocks_per_grid, threads_per_block, 0, stream>>>( d_output, d_input_1, d_input_2, n);
    CHECK_LAST_CUDA_ERROR();

    // 全局内存拷贝到主机内存中
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost, stream));
}

void launch_float_addition_mapped_pinned_memory(float* d_output,
                                                float* d_input_1,
                                                float* d_input_2, uint32_t n,
                                                hipStream_t stream)
{
    // 避免了拷贝, 没有拷贝动作(所以是 零拷贝)
    dim3 const threads_per_block{1024};
    dim3 const blocks_per_grid{32};
    float_addition<<<blocks_per_grid, threads_per_block, 0, stream>>>( d_output, d_input_1, d_input_2, n);
    CHECK_LAST_CUDA_ERROR();
}

void initialize_host_memory(float* h_buffer, uint32_t n, float value)
{
    for (int i{0}; i < n; ++i)
    {
        h_buffer[i] = value;
    }
}

bool verify_host_memory(float* h_buffer, uint32_t n, float value)
{
    for (int i{0}; i < n; ++i)
    {
        if (h_buffer[i] != value)
        {
            return false;
        }
    }
    return true;
}

int main()
{
    constexpr int const num_repeats{10};
    constexpr int const num_warmups{10};

    constexpr int const n{1000000};
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    float const v_input_1{1.0f};
    float const v_input_2{1.0f};
    float const v_output{0.0f};
    float const v_output_reference{v_input_1 + v_input_2};

    hipDeviceProp_t prop;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));
    if (!prop.canMapHostMemory)
    {
        throw std::runtime_error{"Device does not supported mapped memory."};
    }

    // 分配页锁定内存(host)
    float *h_input_1, *h_input_2, *h_output;
    // 分配显存(gpu)
    float *d_input_1, *d_input_2, *d_output;

    // 分配零拷贝内存
    float *a_input_1, *a_input_2, *a_output;
    // 获取其在 gpu 上能够访问这个内存的地址
    float *m_input_1, *m_input_2, *m_output;

    CHECK_CUDA_ERROR(hipHostMalloc(&h_input_1, n * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_input_2, n * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_output, n * sizeof(float), hipHostMallocDefault));

    CHECK_CUDA_ERROR(hipMalloc(&d_input_1, n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_input_2, n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, n * sizeof(float)));

    // cudaHostAllocMapped 分配的主机内存是映射的, 允许 GPU 直接进行访问, 减少了主机和设备之间数据拷贝的开销。
    CHECK_CUDA_ERROR(hipHostAlloc(&a_input_1, n * sizeof(float), hipHostMallocMapped));
    CHECK_CUDA_ERROR(hipHostAlloc(&a_input_2, n * sizeof(float), hipHostMallocMapped));
    CHECK_CUDA_ERROR(hipHostAlloc(&a_output, n * sizeof(float), hipHostMallocMapped));

    // 零拷贝内存虽然不需要显式的传递到设备上, 但是设备还不能通过pHost直接访问对应的内存地址
    // 设备需要访问主机上的的零拷贝内存, 需要先获得另一个地址, 这个地址帮助设备访问到主机对应的内存
    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_input_1, a_input_1, 0));
    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_input_2, a_input_2, 0));
    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_output, a_output, 0));

    // 初始化主机内存
    initialize_host_memory(h_input_1, n, v_input_1);
    initialize_host_memory(h_input_2, n, v_input_2);
    initialize_host_memory(h_output, n, v_output);
    launch_float_addition_non_mapped_pinned_memory(h_output, h_input_1, h_input_2, d_output, d_input_1, d_input_2, n, stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    assert(verify_host_memory(h_output, n, v_output_reference));

    // 初始化主机内存就可以啦
    initialize_host_memory(a_input_1, n, v_input_1);
    initialize_host_memory(a_input_2, n, v_input_2);
    initialize_host_memory(a_output, n, v_output);
    launch_float_addition_mapped_pinned_memory(m_output, m_input_1, m_input_2, n, stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    // 直接验证结果
    assert(verify_host_memory(a_output, n, v_output_reference));

    // 测量延迟
    // 这里是绑定函数
    std::function<void(hipStream_t)> function_non_mapped_pinned_memory{
        std::bind(launch_float_addition_non_mapped_pinned_memory, h_output,
                  h_input_1, h_input_2, d_output, d_input_1, d_input_2, n,
                  std::placeholders::_1)};
    std::function<void(hipStream_t)> function_mapped_pinned_memory{
        std::bind(launch_float_addition_mapped_pinned_memory, m_output,
                  m_input_1, m_input_2, n, std::placeholders::_1)};
    // 开始测量
    float const latency_non_mapped_pinned_memory{measure_performance( function_non_mapped_pinned_memory, stream, num_repeats, num_warmups)};
    float const latency_mapped_pinned_memory{measure_performance( function_mapped_pinned_memory, stream, num_repeats, num_warmups)};

    // 打印测量结果
    std::cout << std::fixed << std::setprecision(3)
              << "CUDA Kernel With Non-Mapped Pinned Memory Latency: "
              << latency_non_mapped_pinned_memory << " ms" << std::endl;
    std::cout << std::fixed << std::setprecision(3)
              << "CUDA Kernel With Mapped Pinned Memory Latency: "
              << latency_mapped_pinned_memory << " ms" << std::endl;

    CHECK_CUDA_ERROR(hipFree(d_input_1));
    CHECK_CUDA_ERROR(hipFree(d_input_2));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipHostFree(h_input_1));
    CHECK_CUDA_ERROR(hipHostFree(h_input_2));
    CHECK_CUDA_ERROR(hipHostFree(h_output));
    CHECK_CUDA_ERROR(hipHostFree(a_input_1));
    CHECK_CUDA_ERROR(hipHostFree(a_input_2));
    CHECK_CUDA_ERROR(hipHostFree(a_output));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
}