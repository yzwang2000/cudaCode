
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// 检查 cuda runtime API 结果的函数, 在 release 中无任何操作(学习下这种写法)
inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

const int TILE_DIM = 32;   // 每个线程块处理的矩阵块的大小为  TILE_DIM * TILE_DIM TILE_DIM 也是线程块中 x 轴方向的线程个数
const int BLOCK_ROWS = 8;  // 这个是线程块中 y 轴方向的线程个数
const int NUM_REPS = 100;

// Check errors and print GB/s, 计算有效带宽(矩阵大小的两倍/执行时间, 矩阵大小两倍是指一次用于加载矩阵, 一次用于存储矩阵)
void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
}

// simple copy kernel 全局内存逐行拷贝到全局内存中
// Used as reference case representing best effective bandwidth.
// 一个 8*32 的线程块处理 32*32 的数据块, 全局内存合并读取, 全局内存合并写入
__global__ void copy(float *odata, float *idata)
{
    // x 和 y 是先定位到当前线程处理原始矩阵的第 x 行, 第 y 列
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;  // 数据矩阵每一行的宽度

    // 这个循环是让每个循环重复其迭代的次数, 这里是一个 warp 读取连续的 32*4 个字节, 读取四次, 好的话, 只需要四个内存事务
    // 也可以改成每个线程读取连续的 4 个字节, 然后总共也是四次内存事务(这样效果感觉会更好一些)
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
        odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// copy kernel using shared memory 全局内存逐行拷贝到共享内存中, 再从共享内存逐行拷贝到全局内存中
// Also used as reference case, demonstrating effect of using shared memory.
// 一个 8*32 的线程块, 处理 32*32 数据块, 这里每个块内分配了 32*32 的共享内存  
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];    // 32*32 的共享内存
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)  // 全局内存逐行合并读入, 逐行存入到共享内存中
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();  // 这行其实不需要, 因为每个 warp 只会操作自己负责的那部分数据, 并没有数据冲突

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// naive transpose 全局内存逐行读取, 然后逐列写入. 这里都是找全部矩阵的行和列, 然后再转置。
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
// 逐行从全局内存读取到共享内存, 逐列的从共享内存读取, 逐行的存储到全局内存中
// 从共享内存读取到全局内存中的时候, 存在 bank 冲突。warp0 中 0 号线程读取 bank0, 1号线程读取 bank0, 2号线程读取 bank0
__global__ void transposeCoalesced(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    // x 和 y 是先定位到当前线程处理原始矩阵的第 x 行, 第 y 列
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // 这个也是限制每个线程迭代的次数, 但是每次改变的是基
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}
   

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    // x 和 y 是先定位到当前线程处理原始矩阵的第 x 行, 第 y 列
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


int main(int argc, char **argv)
{
  const int nx = 1024;  // 需要转置的矩阵的行数
  const int ny = 1024;  // 需要转置的矩阵的列数
  const int mem_size = nx*ny*sizeof(float);  // 矩阵所占用的总的字节数

  dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);  // 依据矩阵的大小和分块的大小来设置线程网格的大小
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);     // 线程块的大小

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("\nDevice : %s\n", prop.name);
  printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", 
         nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  checkCuda(hipSetDevice(devId));

  float *h_idata = (float*)malloc(mem_size);
  float *h_cdata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);
  
  float *d_idata, *d_cdata, *d_tdata;
  checkCuda(hipMalloc(&d_idata, mem_size));
  checkCuda(hipMalloc(&d_cdata, mem_size));
  checkCuda(hipMalloc(&d_tdata, mem_size));

  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }
    
  // host, 产生原始矩阵
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_idata[j*nx + i] = j*nx + i;

  // correct result for error checking, host 端计算转置后的结果
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      gold[j*nx + i] = h_idata[i*nx + j];
  
  // device
  checkCuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  float ms;

  // ------------
  // time kernels
  // ------------
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");
  
  // ----
  // copy 
  // ----
  printf("%25s", "copy");
  checkCuda(hipMemset(d_cdata, 0, mem_size));
  // warm up
  copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda(hipEventRecord(stopEvent, 0) );
  checkCuda(hipEventSynchronize(stopEvent) );
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx*ny, ms);

  // -------------
  // copySharedMem 
  // -------------
  printf("%25s", "shared memory copy");
  checkCuda( hipMemset(d_cdata, 0, mem_size) );
  // warm up
  copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx * ny, ms);

  // --------------
  // transposeNaive 
  // --------------
  printf("%25s", "naive transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

  // ------------------
  // transposeCoalesced 
  // ------------------
  printf("%25s", "coalesced transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

  // ------------------------
  // transposeNoBankConflicts
  // ------------------------
  printf("%25s", "conflict-free transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

error_exit:
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipFree(d_tdata) );
  checkCuda( hipFree(d_cdata) );
  checkCuda( hipFree(d_idata) );
  free(h_idata);
  free(h_tdata);
  free(h_cdata);
  free(gold);
}